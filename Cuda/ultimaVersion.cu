#include "hip/hip_runtime.h"
/*
 * Simulacion simplificada de bombardeo de particulas de alta energia
 *
 * Computacion Paralela (Grado en Informatica)
 * 2017/2018
 *
 * (c) 2018 Arturo Gonzalez Escribano
 */
 #include<stdio.h>
 #include<stdlib.h>
 #include<math.h>
 #include<hip/hip_runtime.h>
 #include<cputils.h>
 
 #define PI	3.14159f
 #define UMBRAL	0.001f
 
 /* Estructura para almacenar los datos de una tormenta de particulas */
 typedef struct {
     int size;
     int *posval;
 } Storm;
 
 
 /* ESTA FUNCION PUEDE SER MODIFICADA */
 /* Funcion para actualizar una posicion de la capa */
 __global__ void actualiza( float *layer, int pos,float energia , int tam ) {
    //  long x = blockDim.y * blockIdx.y + threadIdx.y;
    //  long y = blockDim.x * blockIdx.x + threadIdx.x;
    //  long index = x * gridDim.x + y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
     /* 1. Calcular valor absoluto de la distancia entre el
         punto de impacto y el punto k de la capa */
    if(index< tam){
    int distancia = pos - index;
     if ( distancia < 0 ) distancia = - distancia;
 
     /* 2. El punto de impacto tiene distancia 1 */
     distancia = distancia + 1;
 
     /* 3. Raiz cuadrada de la distancia */
     //float atenuacion = (float)distancia*distancia;
     //float atenuacion = (float)distancia / PI;
     float atenuacion = sqrtf( (float)distancia );
 
     /* 4. Calcular energia atenuada */
     float energia_k = energia / atenuacion;
 
     /* 5. No sumar si el valor absoluto es menor que umbral */
     if ( energia_k >= UMBRAL )
         layer[index] = layer[index] + energia_k;
    }
 }

 __global__ void relaja(float *layer, float *layer_copy, int tam){
    // long x = blockDim.y * blockIdx.y + threadIdx.y;
    // long y = blockDim.x * blockIdx.x + threadIdx.x;
    // long k = x * gridDim.x + y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(0 < k && k< tam-1)
    layer[k] = ( layer_copy[k-1] + layer_copy[k] + layer_copy[k+1] ) / 3;
 }

 __global__ void copia(float *layer, float *layer_copy, int tam){
    // long x = blockDim.y * blockIdx.y + threadIdx.y;
    // long y = blockDim.x * blockIdx.x + threadIdx.x;
    // long k = x * gridDim.x + y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k< tam)
    layer_copy[k] = layer[k];
 }
 __global__ void cMax(float *layer, float *maximos, int *posiciones, int layer_size, int i){
    extern __shared__ float sdata[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(0 < index && index < layer_size - 1){
        sdata[threadIdx.x*2] = layer[index];
        sdata[threadIdx.x*2+1] = (int) index;
    }else{
        sdata[threadIdx.x*2] = 0.0f;
        sdata[threadIdx.x*2+1] = 0;
    }
    __syncthreads();

    // do reduction in shared mem
    for (int s=1; s < blockDim.x; s*=2){
        int new_index = 2 * s * threadIdx.x;

        if(new_index < blockDim.x){
            // if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
                if ( sdata[new_index*2] > sdata[(new_index+s)*2] ) {
                    sdata[new_index*2] = sdata[(new_index+s)*2]; // Copy max
                    sdata[new_index*2+1] = sdata[(new_index+s)*2+1]; // Copy pos
                }
            // }
        }
        __syncthreads();
    }

    if(index == 0){
        maximos[i] = sdata[0];
        posiciones[i] = (int) sdata[1];
    }

}
 
 /* FUNCIONES AUXILIARES: No se utilizan dentro de la medida de tiempo, dejar como estan */
 /* Funcion de DEBUG: Imprimir el estado de la capa */
 void debug_print(int layer_size, float *layer, int *posiciones, float *maximos, int num_storms ) {
     int i,k;
     if ( layer_size <= 35 ) {
         /* Recorrer capa */
         for( k=0; k<layer_size; k++ ) {
             /* Escribir valor del punto */
             printf("%10.4f |", layer[k] );
 
             /* Calcular el numero de caracteres normalizado con el maximo a 60 */
             int ticks = (int)( 60 * layer[k] / maximos[num_storms-1] );
 
             /* Escribir todos los caracteres menos el ultimo */
             for (i=0; i<ticks-1; i++ ) printf("o");
 
             /* Para maximos locales escribir ultimo caracter especial */
             if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                 printf("x");
             else
                 printf("o");
 
             /* Si el punto es uno de los maximos especiales, annadir marca */
             for (i=0; i<num_storms; i++) 
                 if ( posiciones[i] == k ) printf(" M%d", i );
 
             /* Fin de linea */
             printf("\n");
         }
     }
 }
 
 /*
  * Funcion: Lectura de fichero con datos de tormenta de particulas
  */
 Storm read_storm_file( char *fname ) {
     FILE *fstorm = cp_abrir_fichero( fname );
     if ( fstorm == NULL ) {
         fprintf(stderr,"Error: Opening storm file %s\n", fname );
         exit( EXIT_FAILURE );
     }
 
     Storm storm;	
     int ok = fscanf(fstorm, "%d", &(storm.size) );
     if ( ok != 1 ) {
         fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
         exit( EXIT_FAILURE );
     }
 
     storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
     if ( storm.posval == NULL ) {
         fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
         exit( EXIT_FAILURE );
     }
     
     int elem;
     for ( elem=0; elem<storm.size; elem++ ) {
         ok = fscanf(fstorm, "%d %d\n", 
                     &(storm.posval[elem*2]),
                     &(storm.posval[elem*2+1]) );
         if ( ok != 2 ) {
             fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
             exit( EXIT_FAILURE );
         }
     }
     fclose( fstorm );
 
     return storm;
 }
 
 /*
  * PROGRAMA PRINCIPAL
  */
 int main(int argc, char *argv[]) {
     int i,j,k;
 
     /* 1.1. Leer argumentos */
     if (argc<3) {
         fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
         exit( EXIT_FAILURE );
     }
 
     int layer_size = atoi( argv[1] );
     int num_storms = argc-2;
     Storm storms[ num_storms ];
 
     /* 1.2. Leer datos de storms */
     for( i=2; i<argc; i++ ) 
         storms[i-2] = read_storm_file( argv[i] );
 
     /* 1.3. Inicializar maximos a cero */
     float maximos[ num_storms ];
     int posiciones[ num_storms ];
     for (i=0; i<num_storms; i++) {
         maximos[i] = 0.0f;
         posiciones[i] = 0;
     }
 
     /* 2. Inicia medida de tiempo */
     hipSetDevice(0);
     hipDeviceSynchronize();
     double ttotal = cp_Wtime();
 
     /* COMIENZO: No optimizar/paralelizar el main por encima de este punto */
 
     /* 3. Reservar memoria para las capas e inicializar a cero */

     float *layer = (float *)malloc( sizeof(float) * layer_size );
     float *dlayer;
     hipMalloc((void**)&dlayer,sizeof(float) * layer_size);
     dim3 gridShapeGpuFunc1(layer_size/256+(layer_size%256 ? 1:0), 1);
     dim3 bloqShapeGpuFunc1(256, 1);
     
     
     float *layer_copy;
     hipMalloc((void**)&layer_copy,sizeof(float) * layer_size);
     if ( layer == NULL || layer_copy == NULL ) {
         fprintf(stderr,"Error: Allocating the layer memory\n");
         exit( EXIT_FAILURE );
     }
     for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;
     //hipMemcpy(dlayer,layer, sizeof(float) * layer_size,hipMemcpyHostToDevice);
          
     /* 4. Fase de bombardeos */
     for( i=0; i<num_storms; i++) {
         /* 4.1. Suma energia de impactos */
         /* Para cada particula */
         for( j=0; j<storms[i].size; j++ ) {
             /* Energia de impacto (en milesimas) */
             float energia = (float)storms[i].posval[j*2+1] / 1000;
             /* Posicion de impacto */
             int posicion = storms[i].posval[j*2];
 
             /* Para cada posicion de la capa */
                 /* Actualizar posicion */
            actualiza<<<gridShapeGpuFunc1,bloqShapeGpuFunc1>>>( dlayer, posicion, energia, layer_size );

         }
         //hipMemcpy(layer, dlayer, sizeof(float) * layer_size,hipMemcpyDeviceToHost);
 
         /* 4.2. Relajacion entre tormentas de particulas */
         /* 4.2.1. Copiar valores a capa auxiliar */
        //  for( k=0; k<layer_size; k++ ) 
        //      layer_copy[k] = layer[k];
 
         /* 4.2.2. Actualizar capa, menos los extremos, usando valores del array auxiliar */
        //  for( k=1; k<layer_size-1; k++ )
        //    layer[k] = ( layer_copy[k-1] + layer_copy[k] + layer_copy[k+1] ) / 3;  
        copia<<<gridShapeGpuFunc1,bloqShapeGpuFunc1>>>(dlayer, layer_copy, layer_size);
        relaja<<<gridShapeGpuFunc1,bloqShapeGpuFunc1>>>(dlayer, layer_copy, layer_size);
        hipMemcpy(layer, dlayer, sizeof(float) * layer_size,hipMemcpyDeviceToHost);
        
         /* 4.3. Localizar maximo */
         for( k=1; k<layer_size-1; k++ ) {
			/* Comprobar solo maximos locales */
			if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
				if ( layer[k] > maximos[i] ) {
					maximos[i] = layer[k];
					posiciones[i] = k;
				}
			}
		}
     }
 
     /* FINAL: No optimizar/paralelizar por debajo de este punto */
 
     /* 6. Final de medida de tiempo */
     hipDeviceSynchronize();
     ttotal = cp_Wtime() - ttotal;
 
     /* 7. DEBUG: Dibujar resultado (Solo para capas con hasta 35 puntos) */
     #ifdef DEBUG
     debug_print( layer_size, layer, posiciones, maximos, num_storms );
     #endif
 
     /* 8. Salida de resultados para tablon */
     printf("\n");
     /* 8.1. Tiempo total de la computacion */
     printf("Time: %lf\n", ttotal );
     /* 8.2. Escribir los maximos */
     printf("Result:");
     for (i=0; i<num_storms; i++)
         printf(" %d %f", posiciones[i], maximos[i] );
     printf("\n");
 
     /* 9. Liberar recursos */	
     for( i=0; i<argc-2; i++ )
         free( storms[i].posval );
 
     /* 10. Final correcto */
     return 0;
 }
 